#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <data.h>
#include <gpuerrchk.h>

__global__ void kernel(data *obj, double* outarray, int numobj){
  int id = threadIdx.x + blockDim.x*blockIdx.x;
  obj[blockIdx.x].foo(threadIdx.x,outarray[id]);
}

int main(int argc, char* argv[]){
//test data for constructor
  double x = 2.0;
  int i = 2;
  int num = 10;
//N = number of data objects in the array
  int N=3;
  //data bar(x,i,num);
  data* pbar;
  pbar = (data*)malloc(sizeof(data)*N);
  for(int j=0;j<N;j++){
    data temp(x, j, num);
    pbar[j]=temp;} //each data object must be initialized with different input
  double *array = (double*)malloc(sizeof(double)*num*N);
  double *d_array;
  gpuErrchk(hipMalloc((void**)&d_array, sizeof(double)*num*N));
  data *d_pbar;
  gpuErrchk(hipMalloc(&d_pbar, N*sizeof(data)));
  gpuErrchk(hipMemcpy(d_pbar, pbar, N*sizeof(data), hipMemcpyHostToDevice));


  kernel<<<N,num>>>(d_pbar, d_array, N);

  gpuErrchk(hipMemcpy(array,d_array,sizeof(double)*num*N,hipMemcpyDeviceToHost));

  for(i=0;i<num*N;i++)
    printf("array[%d]=%6.2f\n",i,array[i]);

  free(array);
  gpuErrchk(hipFree(d_array));

  return 0;
}
