#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <data.h>
#include <gpuerrchk.h>

__global__ void kernel(data obj, double* outarray){
  int size = obj.msize;
  int id = (blockIdx.x*blockDim.x+threadIdx.x)%size;
  obj.foo(id,outarray[id%size]);
}

int main(int argc, char* argv[]){
//test data for constructor
  double x = 2.0;
  int i = 2;
  int num = 10;
//N = number of data objects in the array
  int N=2;
  //data bar(x,i,num);
  data** pbar;
  pbar = (data**)malloc(sizeof(data*)*N);
  for(int j=0;j<N;j++)
    pbar[j]=new data(x,i,num); //each data object must be initialized with different input
  double *array = (double*)malloc(sizeof(double)*num);
  double *d_array;
  gpuErrchk(hipMalloc((void**)&d_array, sizeof(double)*num));
  

  kernel<<<16,16>>>(*pbar[0], d_array);
                   //only *pbar[0],namely type data works. Both data*, data** would fail. 
                   //It seems illegal to access host pointer data*, data** on device

  gpuErrchk(hipMemcpy(array,d_array,sizeof(double)*num,hipMemcpyDeviceToHost));

  for(i=0;i<num;i++)
    printf("array[%d]=%6.2f\n",i,array[i]);
  
  free(array);
  gpuErrchk(hipFree(d_array));

  return 0;
}
