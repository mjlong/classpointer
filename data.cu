#include "gpuerrchk.h"
#include "data.h"

data::data(double dinput, int iinput, int size){
  msize = size;
  gpuErrchk(hipMalloc((void**)&dptr,sizeof(double)*size));
  gpuErrchk(hipMalloc((void**)&iptr,sizeof(int)*size));
  double *h_dptr = (double*)malloc(sizeof(double)*size);
  int    *h_iptr = (int*)malloc(sizeof(int)*size);
  for(int j=0;j<size;j++){
    h_dptr[j] = dinput * (double)j;
    h_iptr[j] = iinput * j;
  }
  gpuErrchk(hipMemcpy(dptr,h_dptr,sizeof(double)*size,hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(iptr,h_iptr,sizeof(int)*size,hipMemcpyHostToDevice));
  free(h_dptr);
  free(h_iptr);
}

data::~data(){
}

__device__ void data::foo(int input, double& output){
  output = iptr[input] + dptr[input];
}
